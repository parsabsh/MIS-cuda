#include "helpers.cu"
#include <stdio.h>

int main() {
    // showoff();
    int n = 5;
    Graph* graph = new_graph_host(n, n);
    add_undirected_edge(graph, 0, 1);
    add_undirected_edge(graph, 4, 3);
    add_undirected_edge(graph, 3, 2);
    add_undirected_edge(graph, 1, 3);
    // Graph* g_dev = deep_copy_to_device(g);

    // ----------------Deep Copy----------------------

    Graph *dev_graph;
    hipMalloc(&dev_graph, sizeof(Graph));
    hipMemcpy(dev_graph, graph, sizeof(Graph), hipMemcpyHostToDevice);

    Vertex *v;
    hipMalloc(&v, graph->n * sizeof(Vertex));

    hipMemcpy(&(dev_graph->V), &v, sizeof(Vertex *), hipMemcpyHostToDevice);
    
    for (int i = 0; i < graph->n; i++) {
        hipMemcpy(&(v[i]), &(graph->V[i]), sizeof(Vertex), hipMemcpyHostToDevice);
        int *neighbors;
        hipMalloc(&neighbors, graph->max_degree * sizeof(int));
        hipMemcpy(neighbors, graph->V[i].Neighbors, graph->max_degree * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&(v[i].Neighbors), &neighbors, sizeof(int *), hipMemcpyHostToDevice);
    }
    // --------------------------------------


    int* Flags = (int*) malloc(graph->n * sizeof(int));
    // for (int i = 0; i < n; i++)
    // {
    //     Flags[i] = 2;
    // }
    
    memset(Flags, 0, graph->n * sizeof(int));
    int* Flags_dev;
    hipMalloc(&Flags_dev, graph->n * sizeof(int));
    hipMemcpy(Flags_dev, Flags, graph->n * sizeof(int), hipMemcpyHostToDevice);


    int* locks = (int*) malloc(graph->n * sizeof(int));
    memset(locks, 0, graph->n * sizeof(int));
    int* locks_dev;
    hipMalloc(&locks_dev, graph->n * sizeof(int));
    hipMemcpy(locks_dev, locks, graph->n * sizeof(int), hipMemcpyHostToDevice);

    int* lock = (int *) malloc(sizeof(int));
    *lock = 0;
    int *lock_dev;
    hipMalloc(&lock_dev, sizeof(int));
    hipMemcpy(locks_dev, locks, sizeof(int), hipMemcpyHostToDevice);


    // dim3 thread_per_block(1024);
    // dim3 block_per_grid((graph->n + thread_per_block.x - 1) / thread_per_block.x);

    // printf("%d %d\n", thread_per_block, block_per_grid);

    // Lock lock;

    // print_graph_dev<<<block_per_grid, thread_per_block>>>(lock, dev_graph);

    // hipDeviceSynchronize();
    // print_graph(graph);

    size_t blockSize = 256;
    size_t gridSize = (n + blockSize - 1) / blockSize;

    gridSize = n;
    blockSize = 1;

    // maximalIndependentSetKernel<<<gridSize, blockSize>>>(dev_graph, Flags_dev, locks_dev);
    maximalIndependentSet<<<gridSize, blockSize>>>(dev_graph, lock_dev, Flags_dev);
    hipDeviceSynchronize();

    hipMemcpy(Flags, Flags_dev, graph->n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d: %d\n", i, Flags[i]);
    }
}